/*
*	Sergio Isaac Mercado Silvano
*	A01020382
*	CUDA - Lab 1
*	12/11/2019
*/

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

#define MIN(a,b)(a<b?a:b)

#define NUM_RECTS 1e9
#define THREADS 256
#define BLOCKS MIN(32,(NUM_RECTS + THREADS - 1)/THREADS)

__global__ void kernel(float width, float *results){
	__shared__ long cache[THREADS]; //Shared array between threads for all blocks

	int i, cacheIndex;
	float acum, mid, height;

	i = threadIdx.x + (blockIdx.x * blockDim.x); //Id for current thread
	cacheIndex = threadIdx.x;

	acum = 0;	//Acumulated results

	//Actual approximation calculations
	while(i < NUM_RECTS){
		mid = (i + 0.5) * width;
		height = 4.0/(1.0 + mid * mid);
		acum += height;

		i+= blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = acum; //Adding the result for current block (cacheIndex)

	__syncthreads(); //Wait for all threads to finish up previous operations (mutex)

	//Reduction for adding up all partial results in every block

	i = blockDim.x/2;

	while(i > 0){
		if(cacheIndex < i){
			cache[cacheIndex] += cache[cacheIndex + i];
		}

		__syncthreads(); //Same as previous syncthreads()
		i /= 2;
	}

	//End Reduction

	if(cacheIndex == 0){ //Base case
		results[blockIdx.x] = cache[cacheIndex]; //Store results in corresponding index
	}
}

int main(void){
	float acum, width, area;
	float *results, *dr;
	int i;

	width = 1.0/ (double) NUM_RECTS;

	/*
	*	Malloc for result array where we will save all the calculated values in
	*	kernel call
	*/
	results = (float*) malloc(BLOCKS * sizeof(float));

	/*
	*	cudaMalloc for working array that will let us work with the kernel
	*	environment
	*/
	hipMalloc((void**) &dr, BLOCKS * sizeof(float));



	kernel<<<BLOCKS,THREADS>>> (width, dr); //kernel call for GPU calculation


  /*
  * cudaMemcpy for storing the results from the dr* array (which was used)
  * in the kernel to store the results into the results* array for further use
  */
	hipMemcpy(results, dr, BLOCKS * sizeof(float), hipMemcpyDeviceToHost);

	/*
	*	CPU operation for adding every value stored in the previously calculated
	* results from the kernel call to an acumulator variable.
	*/

	acum = 0;
	for(i = 0; i < BLOCKS; i++){
		acum += results[i];
	}

	area = width * acum; //Final calculation for the area (pi)

	printf("PI = %.15lf\n", area);


	//IMPORTANT: Free up device and host memory

	hipFree(dr);
	free(results);

	return 0;
}

//
