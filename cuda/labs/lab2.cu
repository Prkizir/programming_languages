/*
*	Sergio Isaac Mercado Silvano
*	A01020382
*	CUDA - Lab 2
*	19/11/2019
*/

#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 10000

__global__ void kernel(float * a, float * b, float * c, int n){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float temp = 0.0;

  while(row < n && col < n){
    for(int i = 0; i < n; i++){
      temp += a[row * n + i] * b[i * n + col];
    }
  }

  c[row * n + col] = temp;
}

void fill(float * arr, int n, float val){
  for(int i = 0; i < n; i++){
    arr[i] = val/(float)i;
  }
}

void display(float * arr, int n){
  for(int i = 0; i < n; i++){
    printf("%5f ", arr[i]);
  }
}

int main(void){
  float * a, *b, *c;
  float *d_a, *d_b, *d_c;

  a = (float *)malloc(sizeof(float) * N * N);
  b = (float *)malloc(sizeof(float) * N * N);
  c = (float *)malloc(sizeof(float) * N * N);

  hipMalloc((void**) &d_a, sizeof(float) * N * N);
  hipMalloc((void**) &d_b, sizeof(float) * N * N);
  hipMalloc((void**) &d_c, sizeof(float) * N * N);

  fill(a, N * N, 1.0);
  fill(b, N * N, 2.0);

  dim3 threadsPerBlock(N,N);
  dim3 blocksPerGrid(1,1);

  hipMemcpy(d_a, &a, N * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, N * N * sizeof(float), hipMemcpyHostToDevice);

  kernel <<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_c, N * N);

  hipMemcpy(c, &d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

  display(c, N * N);

  free(a);
  free(b);
  free(c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
