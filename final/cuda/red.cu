#include "hip/hip_runtime.h"
/*
* Sergio Mercado A01020382
* Cuda : normal to soviet image
*/


	#include <stdlib.h>
	#include <stdio.h>
	#include <string>
	#include <opencv/highgui.h>
	//#include "utils/cheader.h"


	typedef enum color {BLUE, GREEN, RED} Color;


	__global__ void red(unsigned char *src, unsigned char *dest, int width, int height, int step, int channels){
	  int ren, col;
		float r, g, b;

		ren = blockIdx.x;
		col = threadIdx.x;
		r = 0; g = 0; b = 0;

		r = (float) src[(ren * step) + (col * channels) + RED];
		g = (float) src[(ren * step) + (col * channels) + GREEN];
		b = (float) src[(ren * step) + (col * channels) + BLUE];

		//Set only the desired rgb value
		dest[(ren * step) + (col * channels) + RED] =  (unsigned char) (0xFF);
		dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) (g);
		dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) (b);
	}


	int main(int argc, char* argv[]) {
		int step, size;
		unsigned char *dev_src, *dev_dest;

		//obtain image from source
		IplImage *src = cvLoadImage(argv[1], CV_LOAD_IMAGE_COLOR);
		IplImage *dest = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);

		size = src->width * src->height * src->nChannels * sizeof(uchar);
		step = src->widthStep / sizeof(uchar);

		//Memory allocation on gpu
		hipMalloc((void**) &dev_src, size);
		hipMalloc((void**) &dev_dest, size);

		hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);


		//Write to file
		red<<<src->height, src->width>>>(dev_src, dev_dest, src->width, src->height, step, src->nChannels);
		hipMemcpy(dest->imageData, dev_dest, size, hipMemcpyDeviceToHost);
	  cvSaveImage("result.png", dest);

		hipFree(dev_dest);
		hipFree(dev_src);

		return 0;
	}
